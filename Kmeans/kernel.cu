#include "hip/hip_runtime.h"
#pragma once
#include "Device.h"
#ifndef __HIPCC__   
#define __HIPCC__
#endif
#include <hip/device_functions.h>


int getThreadsPerBlock()
{
	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	return properties.maxThreadsPerBlock;
}

void checkCuda(const hipError_t *const status, int line)
{
	if (*status != hipSuccess)
	{
		printf("CUDA ERROR [line: %d]: %s\n", line - 1, hipGetErrorString(*status));
	}
}

// -- Find optimal number of Blocks& threads 
void cuda_config(unsigned int * gridSize, unsigned int *blockSize, unsigned int dataSize)
{
	unsigned int remainder ;
	unsigned int nThreadPerBlock = getThreadsPerBlock();

	*blockSize = nThreadPerBlock;
	*gridSize = dataSize / nThreadPerBlock;
	remainder = dataSize % nThreadPerBlock;
	if (remainder != 0)
		*gridSize=*gridSize+1;
}

//-- Find the largest diameter between all distances in the current cluster: activate only half of threads -- 
__global__ void get_max(double * distances, unsigned int size) {

	double dist;
	unsigned int index = threadIdx.x + (blockDim.x * blockIdx.x);
	unsigned int totalSize = size;

	while (totalSize > 1)
	{
		unsigned int halfPoint = totalSize / 2;
		unsigned int remained = totalSize % 2;

		if (index < halfPoint || index == halfPoint && remained)
		{
			dist = distances[index + halfPoint];
			if (dist > distances[index]) {
				distances[index] = dist;
			}
		}
		__syncthreads();
		totalSize = totalSize / 2 + remained;
	}
}

//-- Reset all points by time -- 
__global__ void movePoints(Point* points, int N, double dT)
{
	unsigned index = threadIdx.x + blockDim.x *blockIdx.x;
	if (index < N)
	{
		points[index].ID = index;
		points[index].x += points[index].Vx * dT;
		points[index].y += points[index].Vy * dT;
		points[index].z += points[index].Vz * dT;
		points[index].dist = 0;
	}
}

hipError_t cuda_calcDiameter(double* host_distances, unsigned int size)
{
	hipError_t status;
	double* dev_distances;
	unsigned int gridSize, blockSize; 
	
	//-- Allocate device memmory--
	status= hipMalloc((void**)&dev_distances, size * sizeof(double));
	checkCuda(&status, __LINE__);

	//--Copy from host to device memmory --
	status=hipMemcpy(dev_distances, host_distances, size * sizeof(double), hipMemcpyHostToDevice);
	checkCuda(&status, __LINE__);
	
	//-- Set cuda blocks & threads configuration for kernal call
	cuda_config(&gridSize, &blockSize, size);
	get_max<<<gridSize, blockSize>>>(dev_distances, size);
	
	status = hipGetLastError();
	checkCuda(&status, __LINE__);

	//-- Check if any error caused during the kernal launch
	status = hipDeviceSynchronize();
	checkCuda(&status, __LINE__);

	//-- Copy back from device to host memmory -- 
	status = hipMemcpy(host_distances, dev_distances, size * sizeof(double), hipMemcpyDeviceToHost);
	checkCuda(&status, __LINE__);

	//-- Free device memmory
	status= hipFree(dev_distances);
	checkCuda(&status, __LINE__);

	return status;
}

hipError_t cuda_resetPoints(Point* points, unsigned int N, double dT)
{
	Point* dev_points;
	hipError_t status;
	unsigned int gridSize, blockSize;

	status = hipSetDevice(0);
	checkCuda(&status, __LINE__);

	//-- Allocate device memmory--
	status = hipMalloc((void**)&dev_points, N * sizeof(Point));
	checkCuda(&status, __LINE__);

	//--Copy from host to device memmory --
	status = hipMemcpy(dev_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
	checkCuda(&status, __LINE__);
	
	//-- Set cuda blocks & threads configuration for kernal call
	cuda_config(&gridSize, &blockSize, N);
	movePoints <<<gridSize, blockSize >>>(dev_points, N, dT);

	status = hipGetLastError();
	checkCuda(&status, __LINE__);
	
	//-- Check if any error caused during the kernal launch
	status = hipDeviceSynchronize();
	checkCuda(&status, __LINE__);

	//-- Copy back from device to host memmory --  
	status = hipMemcpy(points, dev_points, N * sizeof(Point), hipMemcpyDeviceToHost);
	checkCuda(&status, __LINE__);

	//-- Free device memmory
	hipFree(dev_points);
	return status;
}
